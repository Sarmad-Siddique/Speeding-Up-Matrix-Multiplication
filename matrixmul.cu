
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>
#include <functional>
#include <stdlib.h>    
#include <time.h>       
#include <chrono>

#define ROW_TILE_WIDTH 32
#define COL_TILE_WIDTH 32

template<typename T>
__global__
void matrix_multiply(T *A, T *B, T* C, int width, int C_rows, int C_cols)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;   
    int col = blockIdx.x * blockDim.x + threadIdx.x;
        
    if( row < C_rows && col < C_cols ){
        T value = 0;
        for(int k = 0; k < width; k++){
            value += A[row * width + k] * B[k * C_cols + col];
        }
        C[row * C_cols + col] = value;
    }
}

template<typename T>
void initialize_matrix(T* M, int rows, int cols, std::function<float()> F) {
    for(int i = 0; i < rows; i++){
        for(int j = 0; j < cols; j++){
        M[i * cols + j] = F();
        }
    }
}


int main(void)
{
    int A_rows = 4096;
    int A_cols = 4096;
    int B_rows = 4096;
    int B_cols = 4096;
    int C_rows = 4096;
    int C_cols = 4096;
    int A_size = 4096 * 4096;
    int B_size = 4096 * 4096;
    int C_size = 4096 * 4096;
    float *A, *B, *C;

    
    hipMallocManaged(&A, A_size*sizeof(float));
    hipMallocManaged(&B, B_size*sizeof(float));
    hipMallocManaged(&C, C_size*sizeof(float));

    srand (time(NULL));
    auto rand_numbers = []() -> float {
        auto f = static_cast<float>(rand())/(static_cast<float>(RAND_MAX/1000));
        int n = static_cast<int>(f);
        return static_cast<float>(n);
    };

    initialize_matrix<float>(A, A_rows, A_cols, rand_numbers);
    initialize_matrix<float>(B, B_rows, B_cols, rand_numbers);

    dim3 dim_grid(C_cols/COL_TILE_WIDTH, C_rows/ROW_TILE_WIDTH, 1);
    dim3 dim_block(COL_TILE_WIDTH, ROW_TILE_WIDTH, 1);

    std::cout << "Starting matrix multiplication (GPU): "  << A_rows << " x " << A_cols <<  std::endl;

    uint64_t start_time, end_time, elapsed;

    start_time = std::chrono::duration_cast<std::chrono::milliseconds>(
            std::chrono::high_resolution_clock::now().time_since_epoch())
            .count();
        
    matrix_multiply<float><<<dim_grid, dim_block>>>(A, B, C, A_cols, C_rows, C_cols);

    end_time = std::chrono::duration_cast<std::chrono::milliseconds>(
            std::chrono::high_resolution_clock::now().time_since_epoch())
            .count();

    elapsed = (end_time - start_time);

    std::cout << "ELapsed: " << elapsed << std::endl;

    hipDeviceSynchronize();

    // FILE* fptr = fopen("output.txt", "w");
    	
    // fprintf(fptr, "1\n4096X4096\n");
    	
    // for (int i = 0; i < 4096; i++)
    // {
    // 	for (int j = 0; j < 4096; j++)
    // 	{
    // 		fprintf(fptr, "%f ", C[i * C_cols + j]);
    // 	}
    // 	fprintf(fptr, " \n");
    // }
    	
    // fclose(fptr);

    hipFree(A);
    hipFree(B);
    hipFree(C);
    
    return 0; 
}